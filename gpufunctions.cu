#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

extern "C"

__global__ void vector_add(int n, long long *a, long long *b, long long *c)
{
    int ii = blockIdx.x*blockDim.x + threadIdx.x;
    if(ii < n)
        c[ii] = a[ii] + b[ii];
}

__global__ void addInPlace(int n, long long *oldData, long long *newData)
{
    int ii = blockIdx.x*blockDim.x + threadIdx.x;
    if(ii < n)
        oldData[ii] += newData[ii];
}

__global__ void initMem64_kernel(int n, long long *ptr)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i < n)
        ptr[i] = 0;
}

__global__ void parseAdd_kernel1byte(int numPoints, char *devNewData, long long int *devSum)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i < numPoints)
        devSum[i] += (long long int)devNewData[i];
}

__global__ void parseAdd_kernel2byte(int numPoints, char *devNewData, long long int *devSum, bool le)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i < numPoints)
    {
        if(le)
        {
            int16_t dat = (devNewData[2*i+1] << 8 ) | (devNewData[2*i] & 0xff);
            devSum[i] += (long long int)dat;
        }
        else
        {
            int16_t dat = (devNewData[2*i] << 8 ) | (devNewData[2*i+1] & 0xff);
            devSum[i] += (long long int)dat;
        }
    }
}

namespace GpuAvg {

//device pointers
char *devCharPtr = nullptr;
long long int *devSumPtr = nullptr;


hipError_t gpuFree(void *ptr)
{
    hipError_t err = hipFree(ptr);
    ptr = nullptr;
    return err;
}

hipError_t gpuMalloc(void *ptr, size_t size)
{
    return hipMalloc(&ptr,size);
}

int initializeAcquisition(const int bytesPerPoint, const int numPoints)
{
    hipError_t err;
    if(devSumPtr != nullptr)
    {
        err = gpuFree(devSumPtr);
        if(err != hipSuccess)
            return -1;//QString("Could not free GPU memory for 64 bit data. CUDA error message: %1").arg(QString(cudaGetErrorString(err)));
    }

    err = hipMalloc(&devSumPtr,numPoints*sizeof(long long int));
    if(err != hipSuccess)
    {
        if(devSumPtr != nullptr)
            devSumPtr = nullptr;
        return -2;//QString("Could not allocate GPU memory for 64 bit data. CUDA error message: %1").arg(cudaGetErrorString(err));
    }

    initMem64_kernel<<<(numPoints+255)/256, 256>>>(numPoints,devSumPtr);
    err = hipGetLastError();
    if(err != hipSuccess)
    {
        gpuFree(devSumPtr);
        return -3;//QString("Could not initialize GPU memory to 0 for 64 bit data. CUDA error message: %1").arg(cudaGetErrorString(err));
    }

    if(devCharPtr != nullptr)
    {
        err = gpuFree(devCharPtr);
        if(err != hipSuccess)
        {
            gpuFree(devSumPtr);
            return -4;//QString("Could not free GPU memory for 64 bit data. CUDA error message: %1").arg(QString(cudaGetErrorString(err)));
        }
    }

    err = hipMalloc(&devCharPtr,numPoints*bytesPerPoint*sizeof(char));
    if(err != hipSuccess)
    {
        if(devCharPtr != nullptr)
            devCharPtr = nullptr;
        gpuFree(devSumPtr);
        return -5;//QString("Could not allocate GPU memory for 64 bit data. CUDA error message: %1").arg(QString(cudaGetErrorString(err)));
    }

    return 0;//QString();

}

int acquisitionComplete()
{
    hipError_t err;
//    QString out;
    int out = 0;
    err = gpuFree(devSumPtr);
    if(err != hipSuccess)
        out -=1;// QString("Could not free GPU memory for 64 bit data. CUDA error message: %1").arg(QString(cudaGetErrorString(err)));

    err = gpuFree(devCharPtr);
    if(err != hipSuccess)
    {
        out -=2;
//        if(out.isEmpty())
//            out = QString("Could not free GPU memory for character data. CUDA error message: %1").arg(QString(cudaGetErrorString(err)));
//        else
//            out.append(QString(". Could not free GPU memory for character data. CUDA error message: %1").arg(QString(cudaGetErrorString(err))));
    }

    return out;
}

int gpuParseAndAdd(int bytesPerPoint, int numPoints, const char *newDataIn, long long int *sumData, bool littleEndian = true)
{
    //copy new data to device, run kernel, copy sum from device
    //note: in the future, can try streams and stuff
    hipError_t err;
    err = hipMemcpy(devCharPtr, newDataIn, numPoints*bytesPerPoint*sizeof(char), hipMemcpyHostToDevice);
    if(err != hipSuccess)
        return -1;//QString("Could not copy scope data to GPU. CUDA error message: %1").arg(QString(cudaGetErrorString(err)));


    if(bytesPerPoint == 1)
        parseAdd_kernel1byte<<<(numPoints+255)/256, 256>>>(numPoints,devCharPtr,devSumPtr);
    else
        parseAdd_kernel2byte<<<(numPoints+255)/256, 256>>>(numPoints,devCharPtr,devSumPtr,littleEndian);

    err = hipGetLastError();
    if(err != hipSuccess)
        return -2;//QString("Could not parse and add scope data on GPU. CUDA error message: %1").arg(QString(cudaGetErrorString(err)));

    err = hipMemcpy(sumData, devSumPtr, numPoints*sizeof(long long int), hipMemcpyDeviceToHost);
    if(err != hipSuccess)
        return -3;//QString("Could not copy summed data from GPU. CUDA error message: %1").arg(QString(cudaGetErrorString(err)));

    return 0;//QString();

}

}
